#include "hip/hip_runtime.h"
#include<fstream>
#include<stdio.h>
#include<iostream>
#include "random.h"
long long int  read_file_to_memmory(FILE *pInfile , int *pPointer)
{
    if(pInfile != NULL)
    {
        
        int mIndex =0;
        int mSize = fread(pPointer+mIndex,1,sizeof(int),pInfile);
        long long int mFileSize=0;
        while(mSize!= 0)
        {
            mFileSize = mFileSize +mSize;
            ++mIndex;
            mSize = fread(pPointer+mIndex,1,mSize,pInfile);
        }
        return mFileSize;
    }
    return 0;
}
long long int write_file_from_memmory(FILE *pOutFile , int *pPointer,long long int pFileSize)
{
    if(pOutFile!=NULL)
    {
        pFileSize = fwrite(pPointer,1,pFileSize,pOutFile);
        return pFileSize;
    }
    return 0;
}
long long int generate_random_bits(int  *pPointer , long long int pSize)
{
    long long int mSize = pSize;
    long long int mIndex =0;
    while(pSize>0)
    {
        (*(pPointer+mIndex)) = rand();
        ++mIndex;
        pSize = pSize - sizeof(int);
    }
    return mSize;
}
/*long long int generate_encrypted(int *pDataPointer , int *pRandomData, int *pEncryptedData, long long int pSize)
{
    long long int mSize = pSize;
    long long int mIndex =0;
    while(pSize>0)
    {
        (*(pEncryptedData+mIndex)) = (*(pDataPointer+mIndex))^(*(pRandomData+mIndex));
        ++mIndex;
        pSize = pSize - sizeof(int);
    }
    return mSize;
}*/
__global__ void generate_encrypted(int *pDataPointer , int *pRandomData , int *pEncryptedData , long long int pSize)
{
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    if( index <=(pSize /sizeof(int) ))
    {
        (*(pEncryptedData+index)) = (*(pDataPointer+ index))^(*(pRandomData+index));
    }
    else
        return;
}
int main(int argc , char *argv[])
{
    FILE *inFile;
    FILE *outFile;
    FILE *keyFile;
    inFile = fopen(argv[1],"rb");
    outFile = fopen("enc","wb");
    keyFile = fopen("key","wb");
    int *dataPointer = new int[268435456];
    long long int fileSize = read_file_to_memmory(inFile,dataPointer);
    std::cout<<"yo";
    std::cout<<fileSize;
    int *randomBytePointer = new int[fileSize/sizeof(int) + 100];
    fileSize = generate_random_bits(randomBytePointer , fileSize);
    std::cout<<"hello2";
    int *encryptedPointer = new int[fileSize/sizeof(int) +100];
    int *d_dataPointer;
    int *d_randomBytePointer;
    int *d_EncryptedData;
    hipMalloc((void**)&d_dataPointer,fileSize);
    hipMalloc((void**)&d_randomBytePointer,fileSize);
    hipMalloc((void**)&d_EncryptedData ,fileSize);
    hipMemcpy(d_dataPointer,dataPointer,fileSize,hipMemcpyHostToDevice);
    hipMemcpy(d_randomBytePointer,randomBytePointer,fileSize,hipMemcpyHostToDevice);
    generate_encrypted<<<1000,64>>>(d_dataPointer,d_randomBytePointer,d_EncryptedData,fileSize);
    hipMemcpy(encryptedPointer,d_EncryptedData,fileSize,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::cout<<"hello";
    fileSize =write_file_from_memmory(outFile,encryptedPointer,fileSize);
    fileSize =write_file_from_memmory(keyFile,randomBytePointer,fileSize);
    fclose(inFile);
    fclose(outFile);
    fclose(keyFile);
}
