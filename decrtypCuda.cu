
#include <hip/hip_runtime.h>
#include<fstream>
#include<stdio.h>
#include<iostream>

long long int  read_file_to_memmory(FILE *pInfile , int *pPointer)
{
    if(pInfile != NULL)
    {
        
        int mIndex =0;
        int mSize = fread(pPointer+mIndex,1,sizeof(int),pInfile);
        long long int mFileSize=0;
        while(mSize!= 0)
        {
            mFileSize = mFileSize +mSize;
            ++mIndex;
            mSize = fread(pPointer+mIndex,1,mSize,pInfile);
        }
        return mFileSize;
    }
    return 0;
}
long long int write_file_from_memmory(FILE *pOutFile , int *pPointer,long long int pFileSize)
{
    if(pOutFile!=NULL)
    {
        pFileSize = fwrite(pPointer,1,pFileSize,pOutFile);
        return pFileSize;
    }
    return 0;
}
__global__ void generate_decrypted(int *pDataPointer , int *pRandomData , int *pEncryptedData , long long int pSize)
{
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    if( index <=(pSize /sizeof(int) ))
    {
        (*(pEncryptedData+index)) = (*(pDataPointer+ index))^(*(pRandomData+index));
    }
    else
        return;
}
int main(int argc , char *argv[])
{
    FILE *inFile;
    FILE *outFile;
    FILE *keyFile;
    inFile = fopen("enc","rb");
    keyFile = fopen("key","rb");
    outFile = fopen(argv[1],"wb");
    int *encryptedDataPointer = new int[268435456];
    long long int fileSize = read_file_to_memmory(inFile,encryptedDataPointer);
    int *keyDataPointer = new int[fileSize/sizeof(int) +100];
    int *decryptedDataPointer = new int[fileSize/sizeof(int) +100];
    fileSize = read_file_to_memmory(keyFile,keyDataPointer);
    int *d_encryptedDataPointer;
    hipMalloc((void**)&d_encryptedDataPointer,fileSize);
    int *d_keyPointer;
    hipMalloc((void**)&d_keyPointer,fileSize);
    int *d_decryptedDataPointer;
    hipMalloc((void**)&d_decryptedDataPointer,fileSize);
    hipMemcpy(d_encryptedDataPointer,encryptedDataPointer,fileSize,hipMemcpyHostToDevice);
    hipMemcpy(d_keyPointer , keyDataPointer,fileSize,hipMemcpyHostToDevice);
    generate_decrypted<<<fileSize/64 +1 ,64>>>(d_encryptedDataPointer , d_keyPointer , d_decryptedDataPointer ,fileSize);
    hipMemcpy(decryptedDataPointer,d_decryptedDataPointer,fileSize,hipMemcpyDeviceToHost);
    fileSize = write_file_from_memmory(outFile,decryptedDataPointer,fileSize);
    fclose(inFile);
    fclose(outFile);
    fclose(keyFile);
}
